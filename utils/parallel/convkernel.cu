
#include <hip/hip_runtime.h>
#include <stdio.h>

//Some pre-defined arguments
#define TILE_WIDTH 32

// functions for global id calculating
/**
 * @brief calculate the global id of coordinate (n1,n2) in linearized 2-dimensional 
 * matrix based on row-major layout
 * @param n1 coordinate in direction n1
 * @param n2 coordinate in direction n2
 * @param N2 length of the matrix in direction n2
 * @return global id of (n1,n2)
**/
__device__ int global_id_2d(int n1, int n2, int N2){
    return n2 + N2*n1;
}

// functions for global id calculating
/**
 * @brief calculate the global id of coordinate (n1,n2,n3) in linearized 3-dimensional 
 * matrix based on row-major layout
 * @param n1 coordinate in direction n1
 * @param n2 coordinate in direction n2
 * @param n3 coordinate in direction n3
 * @param N2 length of the matrix in direction n2
 * @param N3 length of the matrix in direction n3
 * @return global id of (n1,n2,n3)
**/
__device__ int global_id_3d(int n1, int n2, int n3, int N2, int N3){
    return n3 + N3*global_id_2d(n1,n2,N2);
    //return n3 + N3*(n2 + N2*n1);
}

/**
 * @brief calculate the global id of coordinate (n1,n2,n3,n4) in linearized 4-dimensional 
 * matrix based on row-major layout
 * @param n1 coordinate in direction n1
 * @param n2 coordinate in direction n2
 * @param n3 coordinate in direction n3
 * @param n4 coordinate in direction n4
 * @param N2 length of the matrix in direction n2
 * @param N3 length of the matrix in direction n3
 * @param N4 length of the matrix in direction n4
 * @return global id of (n1,n2,n3,n4)
**/
__device__ int global_id_4d(int n1, int n2, int n3, int n4, int N2, int N3, int N4){
    return n4 + N4*global_id_3d(n1,n2,n3,N2,N3);
    //return n4 + N4*(n3 + N3*(n2 + N2*n1));
}


/**
 * @brief Naive parallel convolution layer without using shared or constant memory. 
 * mode = valid, stride = 1, mask_width = K.
 * @param X input matrix with size [N, C, H, W]
 * @param Masks masks with size [M, C, K, K]
 * @param Y output matrix with size [N, M, H-K+1, W-K+1]
 * @param N number of samples 
 * @param C number of channels of input matrix
 * @param M number of channels of output matrix
 * @param H height of input matrix
 * @param W width of input matrix
 * @param K width of masks 
 * @param W_grid the number of tiled matrix in width direction
 * @return Convolution result filled in Y
**/
__global__ void convLayer_forward_naive(
    float *X, 
    float *Masks, 
    float *Y, 
    const int N, 
    const int C, 
    const int M, 
    const int H, 
    const int W, 
    const int K,
    const int W_grid){

    // output shape of Y
    const int h_y = H-K+1;
    const int w_y = W-K+1; 

    // initialize some parameters
    int c, p, q;
    const int n = blockIdx.x;
    const int m = blockIdx.y;
    // const int h = blockIdx.z / W_grid + threadIdx.y;
    // const int w = blockIdx.z % W_grid + threadIdx.x;
    const int h = (blockIdx.z / W_grid)*blockDim.y + threadIdx.y;
    const int w = (blockIdx.z % W_grid)*blockDim.x + threadIdx.x;

    float acc = 0;
    // for each input channel
    for(c=0; c<C; c++)
        // convolution
        for(p=0; p<K; p++)          // y-direction
            for(q=0; q<K; q++){      // x-direction
                int gid_x = global_id_4d(n, c, h+p, w+q, C, H, W);
                int gid_m = global_id_4d(m, c, p, q, C, K, K);
                acc += X[gid_x] * Masks[gid_m];
            }
    int gid_y = global_id_4d(n, m, h, w, M, h_y, w_y);
    Y[gid_y] = acc;
}


/**
 * @brief Naive parallel convolution layer without using shared or constant memory. 
 * mode = valid, stride = 1, mask_width = K.
 * @param X input matrix with size [N, H, W, C]
 * @param Masks masks with size [K, K, C, M]
 * @param Y output matrix with size [N, H-K+1, W-K+1, M]
 * @param N number of samples 
 * @param C number of channels of input matrix
 * @param M number of channels of output matrix
 * @param H height of input matrix
 * @param W width of input matrix
 * @param K width of masks 
 * @param W_grid the number of tiled matrix in width direction
 * @return Convolution result filled in Y
**/
__global__ void convLayer_forward_naive_channel(
    float *X, 
    float *Masks, 
    float *Y, 
    int N, 
    int C, 
    int M, 
    int H, 
    int W, 
    int K,
    const int W_grid){

    // output shape of Y
    const int h_y = H-K+1;
    const int w_y = W-K+1; 

    // initialize some parameters
    int c, p, q;
    const int n = blockIdx.x;
    const int m = blockIdx.y;
    const int h = blockIdx.z / W_grid + threadIdx.y;
    const int w = blockIdx.z % W_grid + threadIdx.x;

    float acc = 0;
    // for each input channel
    for(c=0; c<C; c++)
        // convolution
        for(p=0; p<K; p++)          // y-direction
            for(q=0; q<K; q++){      // x-direction
                int gid_x = global_id_4d(n, h+p, w+q, c, H, W, C);
                int gid_m = global_id_4d(p, q, c, m, K, C, M);
                acc += X[gid_x] * Masks[gid_m];
            }
    int gid_y = global_id_4d(n, h, w, m, h_y, w_y, M);
    Y[gid_y] = acc;
}


/**
 * @brief parallel convolution layer using shared memory. 
 * mode = valid, stride = 1, mask_width = K.
 * @param X input matrix with size [N, C, H, W]
 * @param Masks masks with size [M, C, K, K]
 * @param Y output matrix with size [N, M, H-K+1, W-K+1]
 * @param N number of samples 
 * @param C number of channels of input matrix
 * @param M number of channels of output matrix
 * @param H height of input matrix
 * @param W width of input matrix
 * @param K width of masks 
 * @param W_grid the number of tiled matrix in width direction
 * @return Convolution result filled in Y
**/
__global__ void convLayer_forward_shared(
    float *X, 
    float *Masks, 
    float *Y, 
    const int C, 
    const int M, 
    const int H, 
    const int W, 
    const int K,
    const int W_grid){

    // the size to be tiled for X matrix
    const int X_tile_width = TILE_WIDTH + K - 1;
    // allocate shared memory, shared memory size defined when invoking the kernel
    // extern __shared__ float shmem[];
    __shared__ float shmem[1321];
    // first part of shared memory is tile of X, 
    // X_tile has size X_tile_width*X_tile_width
    float *X_shared = &shmem[0];
    // second part of shared memory is part of the mask
    // has size K*K
    // float *Mask_shared = &shmem[X_tile_width*X_tile_width];
    float *Mask_shared = &shmem[1296];
    // Mask_shared[24] = 1;

    // output shape of Y
    const int h_y = H-K+1;
    const int w_y = W-K+1; 

    // initialize some parameters
    // int n, m, h0, w0, h_base, w_base, h, w;
    const int n = blockIdx.x;
    const int m = blockIdx.y;
    const int h0 = threadIdx.x;
    const int w0 = threadIdx.y;
    const int h_base = (blockIdx.z / W_grid) * TILE_WIDTH;
    const int w_base = (blockIdx.z % W_grid) * TILE_WIDTH;
    const int h = h_base + h0;
    const int w = w_base + w0;


    float acc = 0;

    int c, i, j, p, q;
    // for each input channel
    // update the shared memory in each iteration
    for(c=0; c<C; c++){

        // copy mask[m,c,:,:] to the shared memory
        // here h0 = threadIdx.x, w0 = threadIdx.y
        if((h0<K) && (w0<K))
            Mask_shared[global_id_2d(h0,w0,K)] = Masks[global_id_4d(m,c,h0,w0,C,K,K)];
        __syncthreads();

        // copy tiled X to the shared memory
        for(i=h; i<(h_base + X_tile_width); i+=TILE_WIDTH)
            for(j=w; j<(w_base + X_tile_width); j+=TILE_WIDTH)
                if(i<H && j<W)
                    X_shared[global_id_2d(i-h_base,j-w_base,X_tile_width)] = X[global_id_4d(n,c,i,j,C,H,W)];
        __syncthreads();

        // convolution
        for(p=0; p<K; p++)
            for(q=0; q<K; q++)
                acc += X_shared[h+p,w+q] * Mask_shared[p,q];
        __syncthreads();
    }
    Y[global_id_4d(n, m, h, w, M, h_y, w_y)] = acc;
}